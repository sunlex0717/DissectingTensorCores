#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include "../../../hw_def/hw_def.h"

// #define SHARED_MEM_SIZE (32 * 1024 / 4) // 32 KB
// Launch only one thread to calcaulte the latency using a pointer-chasing
// array technique
//#define THREADS_NUM 32
// iterate over the array ITERS times

#ifndef ILPconfig
#define ILPconfig 1
#endif

#ifndef ITERS
#define ITERS  (1024 )
#endif


static_assert(ILPconfig<=8,"ILP > 8 is not supported\n");




__global__ void tensr1688_flops(uint64_t *startClk, uint64_t *stopClk, half *a, half *b, float *res,
          uint32_t strid) { // strid set to 0 used to prevent optimization
  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + tid;
  uint32_t warpid = gid / warpSize;

  a = a + warpid * 16*8; // m*k = 16*16
  b = b + warpid * 8*8; // n*k = 8*16
  res = res + warpid * 16*8;// m*n = 16*16

   /** step 1: create register for each thread **/
  half frag_A[4 * ILPconfig]; // two .f16x2 registers, 8 half elements, 
  half frag_B[4 * ILPconfig];  // one .f16x2 registers, 4 half  elements
  float frag_D[8 * ILPconfig]; //result(fp32) 4 f32 registers
  // fake load, we are focusing on mma latency/throughput. So no need to care about loading
  for(int i = 0;i<4 * ILPconfig;i++){
    frag_A[i] = a[i + lane_id()*4]; 
    
  }
  for(int i =0;i<4 * ILPconfig;i++){
    frag_B[i] = b[i + lane_id()*2]; 
  }

  for(int i=0; i< 8*ILPconfig;i++ ){
    frag_D[i] = 0.0f;
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
  float *C = reinterpret_cast<float *>(&frag_D[0]);
  float *D = C;  // D = A*B + D. 

  // float fpuA = frag_A[0];
  // float fpuB = frag_B[0];
  float fpuC = frag_D[0];

  // int intA = threadIdx.x;
  // int intB = threadIdx.x + 1;
  int intC = threadIdx.x + 2;

  uint64_t start = 0;
  uint64_t stop = 0;
  // synchronize all threads
  asm volatile("bar.sync 0;");
  // start timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");
  //#pragma unroll
  for (int j = 0; j < ITERS; ++j) {
  #if 0
    asm volatile(
        "mma.sync.aligned.m8n8k4.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9,%10,%11}, {%12,%13,%14,%15};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
          "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
    ); 
    #endif
    #if (ILPconfig >= 2) 
    asm volatile(
      "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
      "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[4]), "=f"(D[5]), "=f"(D[6]), "=f"(D[7])
      : "r"(A[2]), "r"(A[3]), 
        "r"(B[1]), 
        "f"(C[4]), "f"(C[5]), "f"(C[6]), "f"(C[7])
      ); 
    #endif
    #if (ILPconfig >= 3)
    asm volatile(
      "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
      "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[8]), "=f"(D[9]), "=f"(D[10]), "=f"(D[11])
      : "r"(A[4]), "r"(A[5]), 
        "r"(B[2]), 
        "f"(C[8]), "f"(C[9]), "f"(C[10]), "f"(C[11])
      ); 
    #endif

    #if (ILPconfig >= 4)
    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[12]), "=f"(D[13]), "=f"(D[14]), "=f"(D[15])
        : "r"(A[6]), "r"(A[7]), 
          "r"(B[3]), 
          "f"(C[12]), "f"(C[13]), "f"(C[14]), "f"(C[15])
    ); 
    #endif

    #if (ILPconfig >= 5)

    asm volatile(
      "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
      "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[16]), "=f"(D[17]), "=f"(D[18]), "=f"(D[19])
      : "r"(A[8]), "r"(A[9]), 
        "r"(B[4]), 
        "f"(C[16]), "f"(C[17]), "f"(C[18]), "f"(C[19])
    );
    #endif

    #if (ILPconfig >= 6)
    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[20]), "=f"(D[21]), "=f"(D[22]), "=f"(D[23])
        : "r"(A[10]), "r"(A[11]), 
          "r"(B[5]), 
          "f"(C[20]), "f"(C[21]), "f"(C[22]), "f"(C[23])
    ); 
    #endif

    #if (ILPconfig >= 7)
    asm volatile(
      "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
      "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
      : "=f"(D[24]), "=f"(D[25]), "=f"(D[26]), "=f"(D[27])
      : "r"(A[12]), "r"(A[13]), 
        "r"(B[6]), 
        "f"(C[24]), "f"(C[25]), "f"(C[26]), "f"(C[27])
    ); 
    #endif

    #if (ILPconfig >= 8)
    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[28]), "=f"(D[29]), "=f"(D[30]), "=f"(D[31])
        : "r"(A[14]), "r"(A[15]), 
          "r"(B[7]), 
          "f"(C[28]), "f"(C[29]), "f"(C[30]), "f"(C[31])
    ); 
    #endif
    //#if (ILPconfig > 1)
    __syncwarp();
    //#endif

  }
  // synchronize all threads
  //__syncwarp();
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  // avoid compiler optimization
  for(int i=0; i < 4 * ILPconfig;i++){
    res[i] += frag_D[i]; 

    res[i] += fpuC;
    res[i] += intC;
  }

  //res[0] += fpuC;
  startClk[gid] = start;
  stopClk[gid] = stop;
}


template <class T, class R> 
float tensor1688_max_flops(int THREADS_PER_BLOCK, bool report_fma_bw = false) {
    intilizeDeviceProp(0);
  
    int BLOCKS_NUM = 1;
    int TOTAL_THREADS = THREADS_PER_BLOCK * BLOCKS_NUM;
    int WARP_SIZE = 32;
  
    unsigned total_A_SIZE =
        16*8 * (TOTAL_THREADS / WARP_SIZE) * ILPconfig; // asume one 16x8 matrix per warp
    unsigned total_B_SIZE =
        8*8 * (TOTAL_THREADS / WARP_SIZE) * ILPconfig; // asume one 8*8 matrix per warp
    unsigned total_R_SIZE =
        16*8 * (TOTAL_THREADS / WARP_SIZE) * ILPconfig; // asume one 16x16 matrix per warp
  
    uint64_t *startClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    uint64_t *stopClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    T *data1 = (T *)malloc(total_A_SIZE * sizeof(T));
    T *data2 = (T *)malloc(total_B_SIZE * sizeof(T));
    R *res = (R *)malloc(total_R_SIZE * sizeof(R));
  
    uint64_t *startClk_g;
    uint64_t *stopClk_g;
    T *data1_g;
    T *data2_g;
    R *res_g;
  
    for (uint32_t i = 0; i < 16*8; i++) {
      data1[i] = (T)i;
    }
  
    for (uint32_t i = 0; i < 8*8; i++) {
      data2[i] = (T)i;
    }
  
    gpuErrchk(hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&data1_g, total_A_SIZE * sizeof(T)));
    gpuErrchk(hipMalloc(&data2_g, total_B_SIZE * sizeof(T)));
    gpuErrchk(hipMalloc(&res_g, total_R_SIZE * sizeof(R)));
  
    gpuErrchk(hipMemcpy(data1_g, data1, total_A_SIZE * sizeof(T),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(data2_g, data2, total_B_SIZE * sizeof(T),
                         hipMemcpyHostToDevice));
  
    tensr1688_flops<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
        startClk_g, stopClk_g, data1_g, data2_g, res_g, 0);
    gpuErrchk(hipPeekAtLastError());
  
    gpuErrchk(hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(
        hipMemcpy(res, res_g, total_R_SIZE * sizeof(R), hipMemcpyDeviceToHost));
  
    float mma_bw, fma_bw;
    uint64_t total_time =  //stopClk[0] - startClk[0];
        *std::max_element(&stopClk[0], &stopClk[TOTAL_THREADS]) -
        *std::min_element(&startClk[0], &startClk[TOTAL_THREADS]);

    float fpuFMA = (float)(ITERS * TOTAL_THREADS * 1 * 1 * 1 * 0 ) /
          ((float)total_time);  // max 64FMA/clk/SM on RTX3070Ti

    mma_bw = ((float)(ITERS * TOTAL_THREADS)) / (float)total_time;
    // hmma_bw = ((float)(REPEAT_TIMES * TOTAL_THREADS * SASS_hmma_per_PTX_wmma)) /
    //           (float)total_time;
    fma_bw = ((float)(ITERS * 16 * 8 * 8 * ILPconfig * //0 *
                      (TOTAL_THREADS / WARP_SIZE))) /
             (float)total_time;
  
    // std::cout << "wmma PTX issue bandwidth = " << wmma_bw << "(thread/clk/SM) \n";
    //std::cout << "mma issue bandwidth = " << mma_bw << "(thread/clk/SM)\n";
    std::cout << "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32  latency " << (float)total_time/(float)ITERS << " cycles\n";
    std::cout << "FMA tensor bandwidth = " << fma_bw + fpuFMA << "(FMA/clk/SM)\n";
  
    std::cout << "Total Clk number = " << total_time << "\n";
  
    if (report_fma_bw)
      return fma_bw;
    else
      return mma_bw;
}

int main() {
    //std::vector<int> warps = {1,2,4,6,8,12,16,20,24,28,32};
    std::vector<int> warps = {1,2,4,6,8,12,16,20,24,28,32};
    //std::vector<int> warps = {20,24,28,32};
    intilizeDeviceProp(0);
    // std::cout << "mma1688 FP16 operand, FP32 accumalte:\n";
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 microbenchmark with ILP = " << ILPconfig << std::endl;
    for(auto& e:warps){
        std::cout << "Number of warps = "<< e <<std::endl;
        tensor1688_max_flops<half, float>(32*e);
        std::cout << std::endl;
    }

    // std::cout << "Number of warps = "<< 1 <<std::endl;
    // tensor1688_max_flops<half, float>(32);
    return 0;
  }
  

#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include "../../../hw_def/hw_def.h"

// #define SHARED_MEM_SIZE (32 * 1024 / 4) // 32 KB
// Launch only one thread to calcaulte the latency using a pointer-chasing
// array technique
//#define THREADS_NUM 32
// iterate over the array ITERS times
#ifndef ITERS
#define ITERS  (1024 )
#endif


#ifndef ILPconfig
#define ILPconfig 1
#endif

static_assert(ILPconfig<=8, "ILP > 8 is not implemented\n");


__global__ void mma_ubench(uint64_t *startClk, uint64_t *stopClk, char *a, char *b, int *res,
          uint32_t strid) { // strid set to 0 used to prevent optimization
  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + tid;
  uint32_t warpid = gid / warpSize;

  a = a + warpid * 16*32; // m*k = 16*32
  b = b + warpid * 8*32; // n*k = 8*32
  res = res + warpid * 16*8;// m*n = 16*8

   /** step 1: create register for each thread **/
  char frag_A[16*ILPconfig]; // four int8 registers, 
  char frag_B[8*ILPconfig];  // one .f16x2 registers, 2 half  elements
  int frag_D[4*ILPconfig]; //result(fp32) 2 f32 registers
  // fake load, we are focusing on mma latency/throughput. So no need to care about loading
  for(int i = 0;i<16*ILPconfig;i++){
    frag_A[i] = a[i + lane_id()*16]; 
    
  }
  for(int i =0;i<8*ILPconfig;i++){
    frag_B[i] = b[i + lane_id()*8]; 
    //frag_D[i] = 0.0f;
  }
  for(int i =0;i<4*ILPconfig;i++){
    //frag_B[i] = b[i + lane_id()*4]; 
    frag_D[i] = 0;
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
  int *C = reinterpret_cast<int *>(&frag_D[0]);
  int *D = C;  // D = A*B + D. 


  uint64_t start = 0;
  uint64_t stop = 0;
  // synchronize all threads
  asm volatile("bar.sync 0;");
  // start timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");
  //#pragma unroll
  for (int j = 0; j < ITERS; ++j) {
    asm volatile(
        "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=r"(D[0]), "=r"(D[1]) , "=r"(D[2]), "=r"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]),
          "r"(C[0]), "r"(C[1]) ,"r"(C[2]), "r"(C[3])
    ); 

    #if ILPconfig >= 2
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[4]), "=r"(D[5]) , "=r"(D[6]), "=r"(D[7])
      : "r"(A[4]), "r"(A[5]), "r"(A[6]), "r"(A[7]), 
        "r"(B[2]), "r"(B[3]),
        "r"(C[4]), "r"(C[5]) ,"r"(C[6]), "r"(C[7])
    ); 
    #endif

    #if ILPconfig >= 3
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[8]), "=r"(D[9]) , "=r"(D[10]), "=r"(D[11])
      : "r"(A[8]), "r"(A[9]), "r"(A[10]), "r"(A[11]), 
        "r"(B[4]), "r"(B[5]),
        "r"(C[8]), "r"(C[9]) ,"r"(C[10]), "r"(C[11])
    ); 
    #endif
    #if ILPconfig >= 4
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[12]), "=r"(D[13]) , "=r"(D[14]), "=r"(D[15])
      : "r"(A[12]), "r"(A[13]), "r"(A[14]), "r"(A[15]), 
        "r"(B[6]), "r"(B[7]),
        "r"(C[12]), "r"(C[13]) ,"r"(C[14]), "r"(C[15])
    ); 
    #endif

    #if ILPconfig >= 5
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[16]), "=r"(D[17]) , "=r"(D[18]), "=r"(D[19])
      : "r"(A[16]), "r"(A[17]), "r"(A[18]), "r"(A[19]), 
        "r"(B[8]), "r"(B[9]),
        "r"(C[16]), "r"(C[17]) ,"r"(C[18]), "r"(C[19])
    ); 
    #endif

    #if ILPconfig >= 6
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[20]), "=r"(D[21]) , "=r"(D[22]), "=r"(D[23])
      : "r"(A[20]), "r"(A[21]), "r"(A[22]), "r"(A[23]), 
        "r"(B[10]), "r"(B[11]),
        "r"(C[20]), "r"(C[21]) ,"r"(C[22]), "r"(C[23])
    ); 
    #endif
    #if ILPconfig >= 7
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[24]), "=r"(D[25]) , "=r"(D[26]), "=r"(D[27])
      : "r"(A[24]), "r"(A[25]), "r"(A[26]), "r"(A[27]), 
        "r"(B[12]), "r"(B[13]),
        "r"(C[24]), "r"(C[25]) ,"r"(C[26]), "r"(C[27])
    ); 
    #endif
    #if ILPconfig >= 8
    asm volatile(
      "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=r"(D[28]), "=r"(D[29]) , "=r"(D[30]), "=r"(D[31])
      : "r"(A[28]), "r"(A[28]), "r"(A[30]), "r"(A[31]), 
        "r"(B[14]), "r"(B[15]),
        "r"(C[28]), "r"(C[29]) ,"r"(C[30]), "r"(C[31])
    ); 
    #endif

    __syncwarp();

  }

  // synchronize warps
 
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");//around 1 cycle overhead
  for(int i=0; i < 4*ILPconfig;i++){
    res[i] = frag_D[i]; 

  }

  //res[0] += fpuC;
  startClk[gid] = start;
  stopClk[gid] = stop;
}


template <class T, class R> 
float run(int THREADS_PER_BLOCK, bool report_fma_bw = false) {
    intilizeDeviceProp(0);
  
    int BLOCKS_NUM = 1;
    int TOTAL_THREADS = THREADS_PER_BLOCK * BLOCKS_NUM;
    int WARP_SIZE = 32;
  
    unsigned total_A_SIZE =
        16*32 * (TOTAL_THREADS / WARP_SIZE); // asume one 16x8 matrix per warp
    unsigned total_B_SIZE =
        8*32 * (TOTAL_THREADS / WARP_SIZE); // asume one 8*8 matrix per warp
    unsigned total_R_SIZE =
        16*8 * (TOTAL_THREADS / WARP_SIZE); // asume one 16x16 matrix per warp
  
    uint64_t *startClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    uint64_t *stopClk = (uint64_t *)malloc(TOTAL_THREADS * sizeof(uint64_t));
    T *data1 = (T *)malloc(total_A_SIZE * sizeof(T));
    T *data2 = (T *)malloc(total_B_SIZE * sizeof(T));
    R *res = (R *)malloc(total_R_SIZE * sizeof(R));
  
    uint64_t *startClk_g;
    uint64_t *stopClk_g;
    T *data1_g;
    T *data2_g;
    R *res_g;
  
    for (uint32_t i = 0; i < 16*32; i++) {
      data1[i] = (T)i;
    }
  
    for (uint32_t i = 0; i < 8*32; i++) {
      data2[i] = (T)i;
    }
  
    gpuErrchk(hipMalloc(&startClk_g, TOTAL_THREADS * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&stopClk_g, TOTAL_THREADS * sizeof(uint64_t)));
    gpuErrchk(hipMalloc(&data1_g, total_A_SIZE * sizeof(T)));
    gpuErrchk(hipMalloc(&data2_g, total_B_SIZE * sizeof(T)));
    gpuErrchk(hipMalloc(&res_g, total_R_SIZE * sizeof(R)));
  
    gpuErrchk(hipMemcpy(data1_g, data1, total_A_SIZE * sizeof(T),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(data2_g, data2, total_B_SIZE * sizeof(T),
                         hipMemcpyHostToDevice));
  
    mma_ubench<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
        startClk_g, stopClk_g, data1_g, data2_g, res_g, 0);
    gpuErrchk(hipPeekAtLastError());
  
    gpuErrchk(hipMemcpy(startClk, startClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS * sizeof(uint64_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(
        hipMemcpy(res, res_g, total_R_SIZE * sizeof(R), hipMemcpyDeviceToHost));
  
    float mma_bw, fma_bw;
    uint64_t total_time =
        *std::max_element(&stopClk[0], &stopClk[TOTAL_THREADS]) -
        *std::min_element(&startClk[0], &startClk[TOTAL_THREADS]);

    float fpuFMA = (float)(ITERS * TOTAL_THREADS * 1 * 1 * 1 * 0 ) /
          ((float)total_time);  // max 64FMA/clk/SM on RTX3070Ti

    mma_bw = ((float)(ITERS * TOTAL_THREADS)) / (float)total_time;
    // hmma_bw = ((float)(REPEAT_TIMES * TOTAL_THREADS * SASS_hmma_per_PTX_wmma)) /
    //           (float)total_time;
    fma_bw = ((float)(ITERS * 16 * 8 * 32 * ILPconfig * //0 *
                      (TOTAL_THREADS / WARP_SIZE))) /
             (float)total_time;
  
    // std::cout << "wmma PTX issue bandwidth = " << wmma_bw << "(thread/clk/SM) \n";
    //std::cout << "mma issue bandwidth = " << mma_bw << "(thread/clk/SM)\n";
    std::cout << "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 latency " << (float)total_time/(float)ITERS << " cycles\n";
    std::cout << "FMA tensor bandwidth = " << fma_bw + fpuFMA << "(FMA/clk/SM)\n";
  
    std::cout << "Total Clk number = " << total_time << "\n";
  
    if (report_fma_bw)
      return fma_bw;
    else
      return mma_bw;
}

int main() {
    intilizeDeviceProp(0);
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 microbenchmark with ILP = " << ILPconfig << std::endl;
    for(int i = 1; i <= 32; i = i*2){
        std::cout << "Number of warps = "<< i <<std::endl;
        run<char, int>(32*i);
        std::cout << std::endl;
    }

    // std::cout << "Number of warps = "<< 1 <<std::endl;
    // tensor1688_max_flops<half, float>(32);
    return 0;
  }
  
